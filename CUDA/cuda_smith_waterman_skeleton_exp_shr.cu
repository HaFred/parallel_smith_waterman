#include "hip/hip_runtime.h"
/**
 * Name: HONG Ziyang
 * Student id: 20528735
 * ITSC email: zhongad@connect.ust.hk
 *
*/

#include <iostream>
#include <string>
#include <algorithm>
#include <chrono>
#include <cassert>
#include <fstream>

using namespace std;

#include "cuda_smith_waterman.h"	
/*
 *  You can add helper functions and variables as you wish.
 */
// #define dev_idx(x, y, n) utils::dev_idx(x, y, n)
using namespace utils;

//expanding
__global__ void expand_score(int a_len, int b_len, int *d_score, int *d_local_max, char *d_a, char *d_b, int re_row, int col_amount) { // re_row device dia rank, re_i
	int threadID = blockDim.x * blockIdx.x + threadIdx.x; 		
	int num_threads = blockDim.x * gridDim.x; // for 1D, how many threads & blocks per have
	int re_i = re_row; // realigned i j
	for (int u = threadID; u < col_amount - 2; u += num_threads) {
		int re_j = u + 1; 
		int j = re_j;		
		int i = re_row - j; // i j are original ele index
		// realign the score for coalesced access
		d_score[dev_idx(re_i, re_j, a_len + 1)] = max(0,
							  max(d_score[dev_idx(re_i - 2, re_j - 1, a_len + 1)] + sub_mat(d_a[i - 1], d_b[j - 1]),
							  max(d_score[dev_idx(re_i - 1, re_j, a_len + 1)] - GAP,
							  d_score[dev_idx(re_i - 1, re_j - 1, a_len + 1)] - GAP)));
		// }
		d_local_max[threadID] = max(d_local_max[threadID], d_score[dev_idx(re_i, re_j, a_len + 1)]);	
	}
}

// not doing multi-for loop in the kernel functions, so no need to do _syncthreads()

// keep & shrinking	
__global__ void shrink_score(int a_len, int b_len, int *d_score, int *d_local_max, char *d_a, char *d_b, int re_row, int col_amount) { // d_rerow device dia rank, re_i
	int threadID = blockDim.x * blockIdx.x + threadIdx.x; 		
	int num_threads = blockDim.x * gridDim.x;	
	int re_i = re_row; 
	if (re_row <= b_len) { //keeping
		for (int u = threadID; u < col_amount - 1; u += num_threads) { 
			int re_j = u;	
			int i = col_amount - 1 - u; // i j are original ele index
			int j = re_row - i;		
			// realign the score for coalesced access
			if (re_i == a_len + 1) {
			d_score[dev_idx(re_i, re_j, a_len + 1)] = max(0,
								  max(d_score[dev_idx(re_i - 2, re_j, a_len + 1)] + sub_mat(d_a[i - 1], d_b[j - 1]),
								  max(d_score[dev_idx(re_i - 1, re_j, a_len + 1)] - GAP,
								  d_score[dev_idx(re_i - 1, re_j + 1, a_len + 1)] - GAP)));
			} else {
				d_score[dev_idx(re_i, re_j, a_len + 1)] = max(0,
								  max(d_score[dev_idx(re_i - 2, re_j + 1, a_len + 1)] + sub_mat(d_a[i - 1], d_b[j - 1]),
								  max(d_score[dev_idx(re_i - 1, re_j, a_len + 1)] - GAP,
								  d_score[dev_idx(re_i - 1, re_j + 1, a_len + 1)] - GAP)));					  							  
			}
			d_local_max[threadID] = max(d_local_max[threadID], d_score[dev_idx(re_i, re_j, a_len + 1)]);
		}
	} else { //shrinking
		for (int u = threadID; u < col_amount; u += num_threads) { 
			int re_j = u;	
			int i = a_len - u; // i j are original ele index
			int j = re_row - i;		
			// realign the score for coalesced access
			if (a_len == b_len && re_row == a_len + 1) {
				d_score[dev_idx(re_i, re_j, a_len + 1)] = max(0,
								  max(d_score[dev_idx(re_i - 2, re_j, a_len + 1)] + sub_mat(d_a[i - 1], d_b[j - 1]),
								  max(d_score[dev_idx(re_i - 1, re_j, a_len + 1)] - GAP,
								  d_score[dev_idx(re_i - 1, re_j + 1, a_len + 1)] - GAP)));
			} else {
				d_score[dev_idx(re_i, re_j, a_len + 1)] = max(0,
								  max(d_score[dev_idx(re_i - 2, re_j + 1, a_len + 1)] + sub_mat(d_a[i - 1], d_b[j - 1]),
								  max(d_score[dev_idx(re_i - 1, re_j, a_len + 1)] - GAP,
								  d_score[dev_idx(re_i - 1, re_j + 1, a_len + 1)] - GAP)));
			}								  							  
			d_local_max[threadID] = max(d_local_max[threadID], d_score[dev_idx(re_i, re_j, a_len + 1)]);
		}
	}	
}

// keep & shrinking	with redundant mem free, to avoid OOM
__global__ void free_shrink_score(int a_len, int b_len, int *d_score, int *d_local_max, char *d_a, char *d_b, int re_row, int col_amount) { // d_rerow device dia rank, re_i
	int threadID = blockDim.x * blockIdx.x + threadIdx.x; 		
	int num_threads = blockDim.x * gridDim.x;	
	int re_i = re_row; 
	if (re_row + (a_len + 1 - 2) <= b_len) { //keeping
		for (int u = threadID; u < col_amount - 1; u += num_threads) { 
			int re_j = u;	
			int i = col_amount - 1 - u; // i j are original ele index
			int j = re_row + (a_len + 1 - 2) - i;		
			// realign the score for coalesced access
			if (re_i == a_len + 1 - (a_len + 1 - 2)) {
			d_score[dev_idx(re_i, re_j, a_len + 1)] = max(0,
								  max(d_score[dev_idx(re_i - 2, re_j, a_len + 1)] + sub_mat(d_a[i - 1], d_b[j - 1]),
								  max(d_score[dev_idx(re_i - 1, re_j, a_len + 1)] - GAP,
								  d_score[dev_idx(re_i - 1, re_j + 1, a_len + 1)] - GAP)));
			} else {
				d_score[dev_idx(re_i, re_j, a_len + 1)] = max(0,
								  max(d_score[dev_idx(re_i - 2, re_j + 1, a_len + 1)] + sub_mat(d_a[i - 1], d_b[j - 1]),
								  max(d_score[dev_idx(re_i - 1, re_j, a_len + 1)] - GAP,
								  d_score[dev_idx(re_i - 1, re_j + 1, a_len + 1)] - GAP)));					  							  
			}
			d_local_max[threadID] = max(d_local_max[threadID], d_score[dev_idx(re_i, re_j, a_len + 1)]);
		}
	} else { //shrinking
		for (int u = threadID; u < col_amount; u += num_threads) { 
			int re_j = u;	
			int i = a_len - u; // i j are original ele index
			int j = re_row + (a_len + 1 - 2) - i;		
			// realign the score for coalesced access
			if (a_len == b_len && re_row + (a_len + 1 - 2) == a_len + 1) {
				d_score[dev_idx(re_i, re_j, a_len + 1)] = max(0,
								  max(d_score[dev_idx(re_i - 2, re_j, a_len + 1)] + sub_mat(d_a[i - 1], d_b[j - 1]),
								  max(d_score[dev_idx(re_i - 1, re_j, a_len + 1)] - GAP,
								  d_score[dev_idx(re_i - 1, re_j + 1, a_len + 1)] - GAP)));
			} else {
				d_score[dev_idx(re_i, re_j, a_len + 1)] = max(0,
								  max(d_score[dev_idx(re_i - 2, re_j + 1, a_len + 1)] + sub_mat(d_a[i - 1], d_b[j - 1]),
								  max(d_score[dev_idx(re_i - 1, re_j, a_len + 1)] - GAP,
								  d_score[dev_idx(re_i - 1, re_j + 1, a_len + 1)] - GAP)));
			}								  							  
			d_local_max[threadID] = max(d_local_max[threadID], d_score[dev_idx(re_i, re_j, a_len + 1)]);
		}
	}	
}

__global__ void update_global_max (int *d_global_max, int *d_local_max) {
	int num_threads = blockDim.x * gridDim.x;
/* 	//debug
	int threadID = blockDim.x * blockIdx.x + threadIdx.x; 		
	printf("Report the thread d_local_max = %d from block %d, thread %d\n", d_local_max[threadID] , blockIdx.x, threadIdx.x); */
    if (threadIdx.x == 0){
        *d_global_max = INT_MIN;
        for (int i = 0; i < num_threads; i++){
            if (d_local_max[i] > *d_global_max){
                *d_global_max = d_local_max[i];
            }
        }
    }
}

//debug
/* __global__ void print_d_score (int *d_score, int num_diagonal, int a_len) {
	int threadID = blockDim.x * blockIdx.x + threadIdx.x; 		
	if (threadID == 0) {
		// printf("\t");
		// for (int i = 0; i < a_len + 1; i++) {
			// cout << b[i] << "\t";
		// }
		printf("\n");
		for (int i = 0; i < 20; i++) {
			// cout << a[i - 1] << "\t";
			for (int j = 0; j < a_len + 1; j++) {
						printf("%d\t", d_score[dev_idx(i, j, a_len + 1)]);
			}
			printf("\n");
		}	
	}	
} */
//debug
/* int print_h_score (int *h_score, int num_diagonal, int a_len) {
	std::ofstream outputf("output_cuda.txt", std::ofstream::out);
	// outputf << dist[0];
	// printf("\n");
	outputf << "\n";
	for (int i = 0; i < num_diagonal; i++) {
		// cout << a[i - 1] << "\t";
		for (int j = 0; j < a_len + 1; j++) {
					// printf("%d\t", d_score[dev_idx(i, j, a_len + 1)]);
			outputf << h_score[i * (a_len + 1) + j] << "\t";
		}
		// printf("\n");
		outputf << "\n";
	}	
	outputf << endl;
	return 0;
} */

int smith_waterman(int blocks_per_grid, int threads_per_block, char *a, char *b, int a_len, int b_len) {
	/*
	 *  Please fill in your codes here.
	 */
	dim3 blocks(blocks_per_grid);
	dim3 threads(threads_per_block);
	
	int num_diagonal = b_len + a_len + 1; // original diagonals, aka the re_row length. 0 padding, a_len + 1 & b_len + 1
	int num_threads = blocks_per_grid * threads_per_block;
	
	// realign the score matrix for coalesced access
	// int *score = (int *)malloc(sizeof(int) * (a_len + 1) * (b_len + 1));
	int *h_score = (int*)calloc(num_diagonal * (a_len + 1), sizeof(int));	
	char *d_a, *d_b;
	int *d_score, *d_local_max, *d_global_max, *global_max;	
	global_max = (int*) malloc(sizeof(int));
	hipMalloc(&d_score, sizeof(int) * (num_diagonal * (a_len + 1)));
	hipMalloc(&d_local_max, sizeof(int) * num_threads);
	hipMalloc(&d_global_max, sizeof(int));
	hipMemset(d_local_max, 0, sizeof(int) * num_threads);
	if (a_len > b_len) {
		hipMalloc(&d_a, sizeof(char) * b_len);
		hipMalloc(&d_b, sizeof(char) * a_len);
		hipMemcpy(d_a, b, sizeof(char) * b_len, hipMemcpyHostToDevice);
		hipMemcpy(d_b, a, sizeof(char) * a_len, hipMemcpyHostToDevice);
		int temp = b_len;
		b_len = a_len;
		a_len = temp;

	} else {
		hipMalloc(&d_a, sizeof(char) * a_len);
		hipMalloc(&d_b, sizeof(char) * b_len);
		hipMemcpy(d_a, a, sizeof(char) * a_len, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, sizeof(char) * b_len, hipMemcpyHostToDevice);
	}
    // hipMemcpy(d_a, a, sizeof(char) * a_len, hipMemcpyHostToDevice);
	// hipMemcpy(d_b, b, sizeof(char) * b_len, hipMemcpyHostToDevice);
    hipMemcpy(d_score, h_score, sizeof(int) * (num_diagonal * (a_len + 1)), hipMemcpyHostToDevice);
		
	int re_row = 0; //readability
	int col_amount = 0;
	int shrink_len = 0;
	bool freeFlag = false;
	shrink_len = a_len + 1;
	
	//expanding Phase
	for (re_row = 2; re_row < a_len + 1; re_row++){ // fred: the re_row = dev_j is the rank_dia, row of realign		
		col_amount = re_row + 1;  
		expand_score <<< blocks, threads >>> (a_len, b_len, d_score, d_local_max, d_a, d_b, re_row, col_amount);
	}
	//shrinking Phase
	if (a_len > 11000 && b_len > 11000) { // if necessary free the redundant mem
		int *temp_d_score;
		freeFlag = true;
		shrink_len = 2;
		num_diagonal = num_diagonal - (a_len + 1 - 2);
		hipMalloc(&temp_d_score, sizeof(int) * (num_diagonal * (a_len + 1)));
		hipMemcpy(temp_d_score, d_score + (a_len - 1) * (a_len + 1), sizeof(int) * 2 * (a_len + 1), hipMemcpyDeviceToDevice);
		hipFree(d_score);
		// int *d_score = temp_d_score;
		for (re_row = shrink_len; re_row < num_diagonal; re_row++){	
			if (re_row > b_len - (a_len + 1 - 2))
				col_amount = num_diagonal - re_row;
			else
				col_amount = a_len + 1;
		free_shrink_score <<< blocks, threads >>> (a_len, b_len, temp_d_score, d_local_max, d_a, d_b, re_row, col_amount);
		hipFree(temp_d_score);
		}
	} else { // not freeing the mem
		for (re_row = shrink_len; re_row < num_diagonal; re_row++){	
			if (re_row > b_len)
				col_amount = num_diagonal - re_row;
			else
				col_amount = a_len + 1;
		shrink_score <<< blocks, threads >>> (a_len, b_len, d_score, d_local_max, d_a, d_b, re_row, col_amount);
		}
	}	
	//update
	update_global_max <<< blocks, threads >>> (d_global_max, d_local_max);
	hipMemcpy(global_max, d_global_max, sizeof(int), hipMemcpyDeviceToHost);
	//debug
	// print_d_score <<< blocks, threads >>> (d_score, num_diagonal, a_len);
	// hipMemcpy(h_score, d_score, sizeof(int) * (num_diagonal * (a_len + 1)), hipMemcpyDeviceToHost);
	// print_h_score(h_score, num_diagonal, a_len);
	free(h_score);
	hipFree(d_a);
	hipFree(d_b);
	if (freeFlag == false)
		hipFree(d_score);
	// else {
		// hipFree(d_score);
	// }
	hipFree(d_local_max);
	hipFree(d_global_max);
	 
	return *global_max;
}